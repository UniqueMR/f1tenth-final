#include "hip/hip_runtime.h"
// my_cuda_code.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include "rrt/rrt.hpp"
#include <geometry_msgs/msg/transform_stamped.hpp>
#include <tf2/LinearMath/Quaternion.h>
#include <tf2/LinearMath/Matrix3x3.h>


#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

// Simple kernel that prints "Hello, World!" from the device
__global__ void hello_world_kernel() {
    printf("Hello, World! from CUDA kernel\n");
}

// Wrapper function to be called from C++
extern "C" void launch_hello_world_kernel() {
    // Launch the kernel with one block and one thread
    hello_world_kernel<<<1, 1>>>();

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
}

__global__ void check_collision_kernel(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                       int check_pts_num, double resolution, 
                                       double origin_x, double origin_y, int width, 
                                       int *map_data, bool *collision_flag) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < check_pts_num) {
        // Calculate the increment step for each thread
        double x_incre = (ptb_x - pta_x) / check_pts_num;
        double y_incre = (ptb_y - pta_y) / check_pts_num;

        // Calculate the sampled point coordinates
        double sampled_pt_x = pta_x + i * x_incre;
        double sampled_pt_y = pta_y + i * y_incre;

        // Map the point to grid indices
        int idx_x = static_cast<int>((sampled_pt_x - origin_x) / resolution);
        int idx_y = static_cast<int>((sampled_pt_y - origin_y) / resolution);
        int idx = idx_y * width + idx_x;

        // Check for collision and set the flag
        if (idx_x >= 0 && idx_y >= 0 && idx < width * width) {
            if (map_data[idx] == 100) {
                *collision_flag = true;
            }
        }
    }
}

extern "C" bool check_collision_cuda(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                 int check_pts_num, double origin_x, double origin_y, 
                                 double resolution, int width, int *map_data) {
    // Device pointers
    int *d_map_data;
    bool *d_collision_flag, h_collision_flag = false;

    // Allocate memory on the device
    hipMalloc(&d_map_data, width * width * sizeof(int));
    hipMalloc(&d_collision_flag, sizeof(bool));

    // Copy data to device
    hipMemcpy(d_map_data, map_data, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_collision_flag, &h_collision_flag, sizeof(bool), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int numBlocks = (check_pts_num + blockSize - 1) / blockSize;

    // Launch the kernel
    check_collision_kernel<<<numBlocks, blockSize>>>(pta_x, pta_y, ptb_x, ptb_y, 
                                                     check_pts_num, resolution, 
                                                     origin_x, origin_y, width, 
                                                     d_map_data, d_collision_flag);

    // Copy the result back to host
    hipMemcpy(&h_collision_flag, d_collision_flag, sizeof(bool), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_map_data);
    hipFree(d_collision_flag);

    return h_collision_flag;
}

const uint ranges_sz = 1080;
const uint updated_map_width = 759;
const uint updated_map_height = 844;
const double updated_map_resolution = 0.1;
const double updated_map_origin_x = -27.7;
const double updated_map_origin_y = -12.4;
const double scan_ang_min = -2.35;
const double scan_ang_increment = 0.00435185;

__global__ void update_occupancy_grid_kernel(float* ranges_arr, uint8_t* updated_map_arr, float t_mat[4][4], double look_ahead_dist, int bubble_offset){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < ranges_sz){
        double curr_dist = ranges_arr[idx];
        double curr_ang = scan_ang_min + idx * scan_ang_increment;

        if(!isnan(curr_dist) && !isinf(curr_dist)){
            double curr_x = curr_dist * cosf(curr_ang);
            double curr_y = curr_dist * sinf(curr_ang);

            if(curr_x < look_ahead_dist && curr_y < look_ahead_dist){
                double curr_global_x = t_mat[0][0] * curr_x + t_mat[0][1] * curr_y + t_mat[0][3];
                double curr_global_y = t_mat[1][0] * curr_x + t_mat[1][1] * curr_y + t_mat[1][3];

                int base_idx_x = static_cast<int>((curr_global_x - updated_map_origin_x) / updated_map_resolution);
                int base_idx_y = static_cast<int>((curr_global_y - updated_map_origin_y) / updated_map_resolution);

                for(int i = base_idx_x - bubble_offset; i < base_idx_x + bubble_offset; i++)
                    for(int j = base_idx_y - bubble_offset; j < base_idx_y + bubble_offset; j++)\
                        if(i * updated_map_width + j > 0 && i * updated_map_width + j < updated_map_width * updated_map_height)
                            updated_map_arr[i * updated_map_width + j] = 100;
                    
            }
        }
    }
        
}

#include <geometry_msgs/msg/transform_stamped.hpp>
#include <tf2/LinearMath/Quaternion.h>
#include <tf2/LinearMath/Matrix3x3.h>

void transformStampedToInverseMatrix(geometry_msgs::msg::TransformStamped& transform, float inverse_matrix[4][4]) {
    // Extract translation
    const auto& t = transform.transform.translation;
    tf2::Quaternion q(
        transform.transform.rotation.x,
        transform.transform.rotation.y,
        transform.transform.rotation.z,
        transform.transform.rotation.w
    );
    tf2::Matrix3x3 m(q);

    // Rotation part of the inverse matrix (transpose of rotation matrix)
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            inverse_matrix[i][j] = static_cast<float>(m[j][i]);  // Transpose the rotation part
        }
    }

    // Translation part of the inverse matrix
    inverse_matrix[0][3] = -(inverse_matrix[0][0] * t.x + inverse_matrix[0][1] * t.y + inverse_matrix[0][2] * t.z);
    inverse_matrix[1][3] = -(inverse_matrix[1][0] * t.x + inverse_matrix[1][1] * t.y + inverse_matrix[1][2] * t.z);
    inverse_matrix[2][3] = -(inverse_matrix[2][0] * t.x + inverse_matrix[2][1] * t.y + inverse_matrix[2][2] * t.z);
    inverse_matrix[3][0] = 0.0f;
    inverse_matrix[3][1] = 0.0f;
    inverse_matrix[3][2] = 0.0f;
    inverse_matrix[3][3] = 1.0f;
}


extern "C" void update_occupancy_grid_cuda(
    const sensor_msgs::msg::LaserScan::ConstSharedPtr scan_msg,
    std::shared_ptr<nav_msgs::msg::OccupancyGrid> updated_map,
    geometry_msgs::msg::TransformStamped& transform,
    double look_ahead_dist, int bubble_offset
){
    float* ranges_arr;
    uint8_t* updated_map_arr;
    hipMalloc(&ranges_arr, ranges_sz * sizeof(float));
    hipMalloc(&updated_map_arr, updated_map_height * updated_map_width);

    hipMemcpy(ranges_arr, scan_msg->ranges.data(), ranges_sz * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(updated_map_arr, 0, updated_map_height * updated_map_width * sizeof(uint8_t)); 
    
    dim3 gridDim(ROUND_UP_TO_NEAREST(ranges_sz, 256));
    dim3 blockDim(256);

    float t_mat[4][4];
    transformStampedToInverseMatrix(transform, t_mat);

    update_occupancy_grid_kernel<<<gridDim, blockDim>>>(ranges_arr, updated_map_arr, t_mat, look_ahead_dist, bubble_offset);

    hipMemcpy(updated_map->data.data(), updated_map_arr, updated_map_width * updated_map_height * sizeof(uint8_t), hipMemcpyDeviceToHost);
    
}


