#include "hip/hip_runtime.h"
// my_cuda_code.cu
#include <hip/hip_runtime.h>
#include <cstdio>
#include "rrt/rrt.hpp"
#include <geometry_msgs/msg/transform_stamped.hpp>
#include <tf2/LinearMath/Quaternion.h>
#include <tf2/LinearMath/Matrix3x3.h>


#define ROUND_UP_TO_NEAREST(M, N) (((M) + (N)-1) / (N))

// Simple kernel that prints "Hello, World!" from the device
__global__ void hello_world_kernel() {
    printf("Hello, World! from CUDA kernel\n");
}

// Wrapper function to be called from C++
extern "C" void launch_hello_world_kernel() {
    // Launch the kernel with one block and one thread
    hello_world_kernel<<<1, 1>>>();

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
}

__global__ void check_collision_kernel(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                       int check_pts_num, double resolution, 
                                       double origin_x, double origin_y, int width, 
                                       int *map_data, bool *collision_flag) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < check_pts_num) {
        // Calculate the increment step for each thread
        double x_incre = (ptb_x - pta_x) / check_pts_num;
        double y_incre = (ptb_y - pta_y) / check_pts_num;

        // Calculate the sampled point coordinates
        double sampled_pt_x = pta_x + i * x_incre;
        double sampled_pt_y = pta_y + i * y_incre;

        // Map the point to grid indices
        int idx_x = static_cast<int>((sampled_pt_x - origin_x) / resolution);
        int idx_y = static_cast<int>((sampled_pt_y - origin_y) / resolution);
        int idx = idx_y * width + idx_x;

        // Check for collision and set the flag
        if (idx_x >= 0 && idx_y >= 0 && idx < width * width) {
            if (map_data[idx] == 100) {
                *collision_flag = true;
            }
        }
    }
}

extern "C" bool check_collision_cuda(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                 int check_pts_num, double origin_x, double origin_y, 
                                 double resolution, int width, int *map_data) {
    // Device pointers
    int *d_map_data;
    bool *d_collision_flag, h_collision_flag = false;

    // Allocate memory on the device
    hipMalloc(&d_map_data, width * width * sizeof(int));
    hipMalloc(&d_collision_flag, sizeof(bool));

    // Copy data to device
    hipMemcpy(d_map_data, map_data, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_collision_flag, &h_collision_flag, sizeof(bool), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int numBlocks = (check_pts_num + blockSize - 1) / blockSize;

    // Launch the kernel
    check_collision_kernel<<<numBlocks, blockSize>>>(pta_x, pta_y, ptb_x, ptb_y, 
                                                     check_pts_num, resolution, 
                                                     origin_x, origin_y, width, 
                                                     d_map_data, d_collision_flag);

    // Copy the result back to host
    hipMemcpy(&h_collision_flag, d_collision_flag, sizeof(bool), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_map_data);
    hipFree(d_collision_flag);

    return h_collision_flag;
}

const uint ranges_sz = 1080;
const uint updated_map_width = 759;
const uint updated_map_height = 844;
const double updated_map_resolution = 0.1;
const double updated_map_origin_x = -27.7;
const double updated_map_origin_y = -12.4;
const double scan_ang_min = -2.35;
const double scan_ang_increment = 0.00435185;

__global__ void update_occupancy_grid_kernel(float* ranges_arr, uint8_t* updated_map_arr, float* t_mat, double look_ahead_dist, int bubble_offset){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if(idx < ranges_sz){
        double curr_dist = ranges_arr[idx];
        double curr_ang = scan_ang_min + idx * scan_ang_increment;

        if(!isnan(curr_dist) && !isinf(curr_dist)){
            double curr_x = curr_dist * cosf(curr_ang);
            double curr_y = curr_dist * sinf(curr_ang);
            
            // printf("Thread %d: curr_dist = %f, curr_ang = %f, curr_x = %f, curr_y = %f, look_ahead_dist = %f\n", idx, curr_dist, curr_ang, curr_x, curr_y, look_ahead_dist);
            if(fabs(curr_x) < look_ahead_dist && fabs(curr_y) < look_ahead_dist){

                double curr_global_x = t_mat[0] * curr_x + t_mat[1] * curr_y + t_mat[3];
                double curr_global_y = t_mat[4] * curr_x + t_mat[5] * curr_y + t_mat[7];

                int base_idx_x = static_cast<int>((curr_global_x - updated_map_origin_x) / updated_map_resolution);
                int base_idx_y = static_cast<int>((curr_global_y - updated_map_origin_y) / updated_map_resolution);

                for(int i = base_idx_x - bubble_offset; i < base_idx_x + bubble_offset; i++)
                    for(int j = base_idx_y - bubble_offset; j < base_idx_y + bubble_offset; j++)
                        if(j * updated_map_width + i > 0 && j * updated_map_width + i < updated_map_width * updated_map_height)
                            updated_map_arr[j * updated_map_width + i] = 100;
            }       
        }
    }
}

extern "C" void update_occupancy_grid_cuda(
    const sensor_msgs::msg::LaserScan::ConstSharedPtr scan_msg,
    std::shared_ptr<nav_msgs::msg::OccupancyGrid> updated_map,
    geometry_msgs::msg::TransformStamped& transform,
    double look_ahead_dist, int bubble_offset
){
    float* ranges_arr;
    uint8_t* updated_map_arr;
    hipMalloc(&ranges_arr, ranges_sz * sizeof(float));
    hipMalloc(&updated_map_arr, updated_map_height * updated_map_width);

    hipMemcpy(ranges_arr, scan_msg->ranges.data(), ranges_sz * sizeof(float), hipMemcpyHostToDevice);


    hipMemset(updated_map_arr, 0, updated_map_height * updated_map_width * sizeof(uint8_t)); 
    
    dim3 gridDim(ROUND_UP_TO_NEAREST(ranges_sz, 256));
    dim3 blockDim(256);

    float t_mat[16];
    transformStampedToMatrix(transform, t_mat);
    float* d_t_mat;

    hipMalloc(&d_t_mat, 4 * 4 * sizeof(float));
    hipMemcpy(d_t_mat, t_mat, 4 * 4 * sizeof(float), hipMemcpyHostToDevice);

    update_occupancy_grid_kernel<<<gridDim, blockDim>>>(ranges_arr, updated_map_arr, d_t_mat, look_ahead_dist, bubble_offset);

    hipMemcpy(updated_map->data.data(), updated_map_arr, updated_map_width * updated_map_height * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipFree(ranges_arr);
    hipFree(updated_map_arr);
    hipFree(d_t_mat);
    
}
