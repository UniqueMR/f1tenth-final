// my_cuda_code.cu
#include <hip/hip_runtime.h>
#include <cstdio>

// Simple kernel that prints "Hello, World!" from the device
__global__ void hello_world_kernel() {
    printf("Hello, World! from CUDA kernel\n");
}

// Wrapper function to be called from C++
extern "C" void launch_hello_world_kernel() {
    // Launch the kernel with one block and one thread
    hello_world_kernel<<<1, 1>>>();

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
}

__global__ void check_collision_kernel(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                       int check_pts_num, double resolution, 
                                       double origin_x, double origin_y, int width, 
                                       int *map_data, bool *collision_flag) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < check_pts_num) {
        // Calculate the increment step for each thread
        double x_incre = (ptb_x - pta_x) / check_pts_num;
        double y_incre = (ptb_y - pta_y) / check_pts_num;

        // Calculate the sampled point coordinates
        double sampled_pt_x = pta_x + i * x_incre;
        double sampled_pt_y = pta_y + i * y_incre;

        // Map the point to grid indices
        int idx_x = static_cast<int>((sampled_pt_x - origin_x) / resolution);
        int idx_y = static_cast<int>((sampled_pt_y - origin_y) / resolution);
        int idx = idx_y * width + idx_x;

        // Check for collision and set the flag
        if (idx_x >= 0 && idx_y >= 0 && idx < width * width) {
            if (map_data[idx] == 100) {
                *collision_flag = true;
            }
        }
    }
}

extern "C" bool check_collision_cuda(double pta_x, double pta_y, double ptb_x, double ptb_y, 
                                 int check_pts_num, double origin_x, double origin_y, 
                                 double resolution, int width, int *map_data) {
    // Device pointers
    int *d_map_data;
    bool *d_collision_flag, h_collision_flag = false;

    // Allocate memory on the device
    hipMalloc(&d_map_data, width * width * sizeof(int));
    hipMalloc(&d_collision_flag, sizeof(bool));

    // Copy data to device
    hipMemcpy(d_map_data, map_data, width * width * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_collision_flag, &h_collision_flag, sizeof(bool), hipMemcpyHostToDevice);

    // Define grid and block size
    int blockSize = 256;
    int numBlocks = (check_pts_num + blockSize - 1) / blockSize;

    // Launch the kernel
    check_collision_kernel<<<numBlocks, blockSize>>>(pta_x, pta_y, ptb_x, ptb_y, 
                                                     check_pts_num, resolution, 
                                                     origin_x, origin_y, width, 
                                                     d_map_data, d_collision_flag);

    // Copy the result back to host
    hipMemcpy(&h_collision_flag, d_collision_flag, sizeof(bool), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_map_data);
    hipFree(d_collision_flag);

    return h_collision_flag;
}
