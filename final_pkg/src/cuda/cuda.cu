// my_cuda_code.cu
#include <hip/hip_runtime.h>
#include <cstdio>

// Simple kernel that prints "Hello, World!" from the device
__global__ void hello_world_kernel() {
    printf("Hello, World! from CUDA kernel\n");
}

// Wrapper function to be called from C++
extern "C" void launch_hello_world_kernel() {
    // Launch the kernel with one block and one thread
    hello_world_kernel<<<1, 1>>>();

    // Synchronize to ensure kernel completion
    hipDeviceSynchronize();
}
